#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>

#define PI 3.14
#define EPS 0.0313
#define N 1024*10
#define THREADS_PER_BLOCK 256
#define BLOCK_COUNT 256

hipError_t SortWithCuda(double* array_,int*array_pos, int block_size, int Length);

__global__ void SlimSortKernel(double* array, int ArLength, int BlockSize) {
    int pos =threadIdx.x+ blockIdx.x*blockDim.x;
    if (pos < ArLength) {
        array_pos
   }
}
int main()
{

    double* dat = new double[N];
    double sum = 0,control_sum=0;
    srand(time(0));
    for (int i = 0; i < N; i++) {
        dat[i] = 256 - 512 * sin((PI / (rand() - EPS)) - EPS);
        //sum += dat[i];

    }
    for (int i = 0; i < N; i++)
        sum += dat[i];
    // Add vectors in parallel.
    hipError_t cudaStatus=hipSuccess;
   // cudaStatus= addWithCuda(c, a, b, arraySize);
    cudaStatus = SortWithCuda(dat, 2, N);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        //return 1;
        goto Error;
    }
    bool suc = true;
    ;
    for (int i = 0; i < N-1; i++) {
        control_sum += dat[i];
        if (dat[i] > dat[i + 1]) {
            suc = false;
            break;
        }
    }
    for (int i = 0; i < N; i++)
        control_sum += dat[i];
    if (!suc) {

        printf("error sort!\n");
       
    }
    if (sum != control_sum) {
 printf("{%d}:{%d}", control_sum, sum);
    }
    /*printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);*/

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        //return 1;
        goto Error;
    }
Error:
    delete[]dat;
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.

hipError_t SortWithCuda(double* array_,int block_size,int Length) {
    double* dev_a = 0;
    double* dev_pos_a = 0;

    hipError_t cudaStatus;
    hipStream_t cudaStream = 0;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
    cudaStatus = hipStreamCreate(&cudaStream);
    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_a, Length * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_pos_a, Length * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_a, array_, Length * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    while (block_size<=Length)
    {
        printf("block_size:%d\n", block_size);
        dim3 size(256, 256);
        SlimSortKernel<<<BLOCK_COUNT,THREADS_PER_BLOCK,0,cudaStream>>>(dev_a,dev_pos_a, Length, block_size);
      //  cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            fprintf(stderr, "cuda error%s", hipGetErrorString(cudaStatus));
            goto Error;
        }
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
      
          fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
                fprintf(stderr, "cuda error%s", hipGetErrorString(cudaStatus));
          goto Error;
        }
        
        if (block_size < Length) {
block_size *= 2;
if (block_size > Length) {
    block_size = Length;
}
        }
        else {
            break;
        }
        
    }
    cudaStatus = hipMemcpy(array_, dev_a, Length * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree((void*)dev_a);
    hipStreamDestroy(cudaStream);
    return cudaStatus;
}